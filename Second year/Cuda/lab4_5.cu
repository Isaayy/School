#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <time.h>
#define TILE_WIDTH 4

using namespace std;

__global__ void mul(double* Md, double* Nd, double* Pd, int Width) {

    __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];

    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    double w = 0.0;

    for (int i = 0; i < Width / TILE_WIDTH; ++i) {
        Mds[threadIdx.y][threadIdx.x] = Md[Row * Width + (i * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[Col + (i * TILE_WIDTH + threadIdx.y) * Width];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++)
            w += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        __syncthreads();
    }
    Pd[Row * Width + Col] = w;
}

double losuj(double fMin, double fMax)
{
    double f = (double)rand() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

int main(void) {
    double** x, ** y, ** z;
    double* d_x, * d_y, * d_z;
    int Width = 4;
    int size = Width * Width * sizeof(double);

    // ALOKACJA
    x = new double* [Width];
    y = new double* [Width];
    z = new double* [Width];

    x[0] = new double[Width * Width];
    y[0] = new double[Width * Width];
    z[0] = new double[Width * Width];

    for (int i = 1; i < Width; i++) {
        x[i] = x[0] + i * Width;
        y[i] = x[0] + i * Width;
        z[i] = x[0] + i * Width;
    }

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    // WYPELNIENIE
    srand((unsigned int)time(NULL));
    for (int i = 0; i < Width * Width; i++) {
        x[0][i] = losuj(-1, 1);
        y[0][i] = losuj(-1, 1);
        z[0][i] = 0;
    }

    hipMemcpy(d_x, x[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y[0], size, hipMemcpyHostToDevice);

    dim3 dimGrid(Width / TILE_WIDTH, Width / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    mul << <dimGrid, dimBlock >> > (d_x, d_y, d_z, Width);

    hipMemcpy(z[0], d_z, size, hipMemcpyDeviceToHost);

    // WYPISANIE MACIERZY
    for (int i = 0; i < Width * Width; i++) {
        if (i % Width == 0) cout << endl;
        cout << x[0][i] << " ";
    }
    cout << endl;
    for (int i = 0; i < Width * Width; i++) {
        if (i % Width == 0) cout << endl;
        cout << y[0][i] << " ";
    }
    cout << endl;
    for (int i = 0; i < Width * Width; i++) {
        if (i % Width == 0) cout << endl;
        cout << z[0][i] << " ";
    }

    // ZWOLNIENIE PAMIECI
    delete[] x[0];
    delete[] y[0];
    delete[] z[0];

    delete[] x;
    delete[] y;
    delete[] z;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}