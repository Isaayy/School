#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define RADIUS 3
#define BLOCK_SIZE 64

using namespace std;

__global__ void filtr(double* in, double* out, int N)
{
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    if (gindex >= N || lindex >= N) {
        return;
    }

    __shared__ unsigned int temp[BLOCK_SIZE + 2 * RADIUS];

    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS < 0 ? N + gindex - RADIUS : gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE >= N ? gindex + BLOCK_SIZE - N : gindex + BLOCK_SIZE];
    }

    __syncthreads();

    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
        result += temp[lindex + offset];
    }
    out[gindex] = result;

}

double losuj(double fMin, double fMax)
{
    double f = (double)rand() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

int main()
{
    // ROZMIAR

    int N = 10 + 2 * RADIUS;
    int size = N * sizeof(double);

    double* a, * b;
    double* d_x, * d_y;

    // alokacja pamieci wektorow na HOST
    a = (double*)malloc(size);
    b = (double*)malloc(size);

    // alokacja na GPU
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    // Uzupelnienie wektorow liczbami losowymi na HOST
    for (int i = 0; i < N; i++) {
        a[i] = losuj(-1, 1);
        b[i] = 0;
    }

    hipMemcpy(d_x, a, size, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(N);

    filtr << <blocksPerGrid, BLOCK_SIZE >> > (d_x, d_y, N);

    hipDeviceSynchronize();

    hipMemcpy(b, d_y, size, hipMemcpyDeviceToHost);

    for (int k = 0; k < N; k++)
        cout << a[k] << " ";

    free(a);
    free(b);

    hipFree(d_x);
    hipFree(d_y);

    return 0;
}