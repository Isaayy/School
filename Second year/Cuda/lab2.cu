#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


using namespace std;

__global__ void addKernel(double* x, double* y, double* c, int M, int N)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * M + col;
    if (col < M && row < N)
        c[index] = x[index] + y[index];
}

double losuj(double fMin, double fMax)
{
    double f = (double)rand() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

int main()
{
    double* d_x, * d_y, * d_c;

    // ROZMIARY
    int M = 8;
    int N = 5;

    // ZAINICNJOWANIE MACIERZY
    double** x;
    x = new double* [M];
    double** y;
    y = new double* [M];
    y[0] = new double[M * N];
    x[0] = new double[M * N];

    for (int i = 0; i < M; i++) {
        x[i] = x[0] + i * N;
        y[i] = y[0] + i * N;
    }

    // ZAINICNJOWANIE MACIERZY WYNIKOWEJ
    double** c;
    c = new double* [M];
    c[0] = new double[M * N];
    for (int i = 0; i < M; i++)
        c[i] = c[0] + i * N;

    // UZUPELNIENIE ICH LICZBAMI LOSOWYMI
    for (int i = 0; i < M; i++) {
        for (int k = 0; k < N; k++) {
            x[i][k] = losuj(-1, 1);
            y[i][k] = losuj(-1, 1);
        }
    }

    // alokacja na GPU
    hipMalloc(&d_x, (M * N) * sizeof(double));
    hipMalloc(&d_y, (M * N) * sizeof(double));
    hipMalloc(&d_c, (M * N) * sizeof(double));

    hipMemcpy(d_x, x[0], (M * N) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y[0], (M * N) * sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(M, N); // 2 wymiarowa
    addKernel << <1, threadsPerBlock >> > (d_x, d_y, d_c, M, N); // 1blok 2 wymiarowy // robic na wielu blokach

    hipMemcpy(c[0], d_c, (M * N) * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++) {
        for (int k = 0; k < N; k++) {
            cout << c[i][k];
        }
    }

    delete[] c[0];
    delete[] x[0];
    delete[]x;
    delete[] y[0];
    delete[]y;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_c);


    return 0;
}

