#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define N 512

using namespace std;

__global__ void addKernel(double* a, double* b, double* c, int n)
{
    // int id = blockIdx.x ; //  * blockDim.x + threadIdx.x; // 1 watek wiec thread = 0 

    // Aby nie wyjsc poza granice
    if (id < n)
        c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

double losuj(double fMin, double fMax)
{
    double f = (double)rand() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

int main()
{
    // ROZMIAR

    int size, n;
    // cout << "Podaj rozmiar wektorow ";
    // cin >> n;
    n = 10;
    size = N * sizeof(double);


    double* a, * b, * c;
    double* d_a, * d_b, * d_c;

    // alokacja pamieci wektorow na HOST
    a = (double*)malloc(size);
    b = (double*)malloc(size);
    c = (double*)malloc(size);

    // alokacja na GPU
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    int i;
    // Uzupelnienie wektorow liczbami losowymi na HOST
    for (i = 0; i < n; i++) {
        a[i] = losuj(-1, 1);
        b[i] = losuj(-1, 1);
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    addKernel << < N, 1 >> > (d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int k = 0; k < n; k++)
        cout << c[k] << ' ';

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

