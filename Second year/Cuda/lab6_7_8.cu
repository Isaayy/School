#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <time.h>
#define SIZE 16


using namespace std;


// ##########################################################################################

__global__ void reduce1(int *g_idata, int *g_odata)
{
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid]=g_idata[i];
    __syncthreads();

    for(unsigned int s = 1 ; s < blockDim.x ; s*=2){
        if (tid % (2*s)==0){
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    if (tid == 0 ) g_odata[blockIdx.x]=sdata[0];
}

// ##########################################################################################

__global__ void reduce2(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
 
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
 
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2*s*tid;
 
        if (index <  blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    } 
 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// ##########################################################################################

__global__ void reduce3(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
 
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
 
    for (unsigned int s = blockDim.x/2 ; s > 0 ; s >>=1) {
        if (tid<s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    } 
 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// ##########################################################################################

__global__ void reduce4(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
 
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i]+g_idata[i+blockDim.x];
    __syncthreads();
 
    for (unsigned int s = blockDim.x/2 ; s > 0 ; s >>=1) {
        if (tid<s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    } 
 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


// ##########################################################################################

template<unsigned int blockSize>
__device__ void warpReduce1(volatile int *sdata, int tid){
    if (blockSize >= 64) sdata[tid] += sdata[tid+32];
    if (blockSize >= 32) sdata[tid] += sdata[tid+16];
    if (blockSize >= 16) sdata[tid] += sdata[tid+8];
    if (blockSize >= 8) sdata[tid] += sdata[tid+4];
    if (blockSize >= 4) sdata[tid] += sdata[tid+2];
    if (blockSize >= 2) sdata[tid] += sdata[tid+1];
}
 
template<unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
 
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();
 
    for (unsigned int s = blockDim.x/2; s > 32; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid+s];
        }
 
        __syncthreads();
    } 
 
    if (blockSize >= 512) {
        if (tid < 256) {sdata[tid] += sdata[tid+256];} __syncthreads();}
    if (blockSize >= 256) {
        if (tid < 128) {sdata[tid] += sdata[tid+128];} __syncthreads();}
    if (blockSize >= 128) {
        if (tid < 64) {sdata[tid] += sdata[tid+64];} __syncthreads();}
 
    if (tid < 32) warpReduce1<blockSize>(sdata, tid);
 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


int losuj(int min, int max)
{
    
    return  min + rand() % (( max + 1 ) - min);
}
 
int main(void) {
    
     int numThreadsPerBlock = 1024;
     int *arrayIn;
     int *arrayOut;
     
     int *d_input;
     int *d_output;
      

     int sum = 0 ;

    int numOutputElements; 

    numOutputElements = SIZE / (numThreadsPerBlock / 2);
    if (SIZE % (numThreadsPerBlock / 2)) {
        numOutputElements++;
    }

    arrayIn = (int *)malloc(SIZE * sizeof(int));
    arrayOut = (int *)malloc(numOutputElements * sizeof(int));

    for(int i = 0 ; i< SIZE ; i++){
          arrayIn[i]=losuj(-10, 10); 
          sum+= arrayIn[i];
    }


    hipMalloc((void **)&d_input, SIZE *sizeof(int));
    hipMalloc((void **)&d_output, numOutputElements *sizeof(int));

   
    hipMemcpy(d_input, arrayIn , SIZE*sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(numThreadsPerBlock, 1, 1);
    dim3 gridSize(numOutputElements, 1, 1);

    

    switch (numThreadsPerBlock){
      case 512:
        reduce6<512><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 256:
        reduce6<256><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 128:
        reduce6<128><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 64:
        reduce6<64><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 32:
        reduce6<32><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 16:
        reduce6<16><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 8:
        reduce6<8><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 4:
        reduce6<4><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;
      case 2:
        reduce6<2><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break; 
      case 1:
        reduce6<1><<<gridSize,blockSize,numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);break;                                                                     
    }


    reduce4<<<gridSize, blockSize, numThreadsPerBlock*sizeof(int)>>>(d_input, d_output);


    


    hipMemcpy(arrayOut, d_output, numOutputElements *sizeof(int), hipMemcpyDeviceToHost);
 
    // WYPISANIE TABLICY
    cout << "Index 0 otrzmanej tablicy (suma): " << arrayOut[0] ;


    // WYPISANIE TABLICY device
    cout << endl <<"Tablica host : ";
    for (int i = 0; i < SIZE; i++) {
        cout << arrayIn[i]<< " ";
    }
    cout<<endl<<"Suma obliczona na host: " << sum << endl;
 
    // ZWOLNIENIE PAMIECI
    free(arrayIn);
    free(arrayOut);

    hipFree(d_input);
    hipFree(d_output) ;

 
    return 0;
}